#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
      int x = threadIdx.x;
      int y = threadIdx.y;
      int x_block = blockIdx.x;
      int y_block = blockIdx.y;

      int x_bdim = blockDim.x;
      int y_bdim = blockDim.y;
      int x_gdim = gridDim.x;
      int y_gdim = gridDim.y;

      int x_pos = x_bdim * x_block * x;
      int y_pos = y_bdim * y_block *y;

      int offset =  y_pos * (x_bdim * x_gdim) + x_pos;

      uchar4 rgba = rgbaImage[offset];  //r * numCols + c
      float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
      greyImage[offset] = channelSum;

}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  const dim3 blockSize(numRows/16+1, numCols/16+1, 1); 
  const dim3 gridSize( 16, 16, 1); 
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}